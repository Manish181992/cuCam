#include "hip/hip_runtime.h"
#include <SDL.h>
#include <hip/hip_runtime.h>
#include <>


struct pix24{
	unsigned char r;
	unsigned char g;
	unsigned char b;
};

__global__ void oldtimes(pix24 *pixels){

	pix24 *pixel = &pixels[blockIdx.x * blockDim.x + threadIdx.x];
	float intensity = .299f * pixel->r + .587f * pixel->g + .114f * pixel->b;
	
	pixel->r = intensity;
	pixel->g = intensity;
	pixel->b = intensity;
}

SDL_Surface *manSurf(SDL_Surface *surf, int manType){

	struct pix24 *d_pixels = NULL;
	int noOfPixels = surf->w * surf->h;

	const dim3 gridDims(surf->h,1,1);
	const dim3 blockDims(surf->w,1,1);
	
	hipMalloc((void **)&d_pixels, noOfPixels * (3));
	hipMemcpy(d_pixels, surf->pixels, noOfPixels * 3, hipMemcpyHostToDevice);

	switch(manType){
		case 1:
			oldtimes<<<gridDims, blockDims>>>(d_pixels);
			break;
	};

	hipMemcpy(surf->pixels, d_pixels, noOfPixels * 3, hipMemcpyDeviceToHost);
	hipFree(d_pixels);
	return surf;
}